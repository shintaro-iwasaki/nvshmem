#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define MAX_MSG_SIZE 64 * 1024
#define THREADS_PER_WARP 32
#define THREADS_PER_BLOCK 1024

__global__ void latency(volatile int *data_d, volatile int *flag_d, int len, int pe, int iter,
                        int skip) {
    long long int start, stop;
    double usec, time;
    int i, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < (iter + skip); i++) {
        if (i == skip) {
            nvshmem_quiet();
            start = clock64();
        }

        nvshmem_int_put_nbi((int *)data_d, (int *)data_d, len, peer);

        nvshmem_quiet();
    }
    stop = clock64();

    if (!tid) {
        time = (stop - start) / iter;
        usec = time * 1000 / clockrate;
        printf("%7lu \t %8.2f \n", len * sizeof(int), usec);
    }
}

#define LATENCY_THREADGROUP(group)                                                               \
    __global__ void latency_##group(volatile int *data_d, volatile int *flag_d, int len, int pe, \
                                    int iter, int skip) {                                        \
        long long int start, stop;                                                               \
        double usec, time;                                                                       \
        int i, tid, peer;                                                                        \
                                                                                                 \
        peer = !pe;                                                                              \
        tid = threadIdx.x;                                                                       \
                                                                                                 \
        for (i = 0; i < (iter + skip); i++) {                                                    \
            if (i == skip) {                                                                     \
                __syncthreads();                                                                 \
                if (!tid) {                                                                      \
                    nvshmem_quiet();                                                             \
                    start = clock64();                                                           \
                }                                                                                \
                __syncthreads();                                                                 \
            }                                                                                    \
                                                                                                 \
            nvshmemx_int_put_##group((int *)data_d, (int *)data_d, len, peer);                   \
                                                                                                 \
            __syncthreads();                                                                     \
            if (!tid) nvshmem_quiet();                                                           \
            __syncthreads();                                                                     \
        }                                                                                        \
                                                                                                 \
        if (!tid) {                                                                              \
            stop = clock64();                                                                    \
            time = (stop - start) / iter;                                                        \
            usec = time * 1000 / clockrate;                                                      \
            printf("%7lu \t %8.2f \n", len * sizeof(int), usec);                                  \
        }                                                                                        \
    }

LATENCY_THREADGROUP(warp)
LATENCY_THREADGROUP(block)

int main(int c, char *v[]) {
    int mype, npes, size;
    int *flag_d = NULL, *data_d = NULL;

    int iter = 200;
    int skip = 20;
    int max_msg_size = MAX_MSG_SIZE;

    init_wrapper(&c, &v);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    data_d = (int *)nvshmem_malloc(max_msg_size);
    flag_d = (int *)nvshmem_malloc(sizeof(int));
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));
    CUDA_CHECK(hipMemset(flag_d, 0, sizeof(int)));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        printf("   Put with thread \n");
        printf("   size(bytes) \t latency(us)\n");
        fflush(stdout);
    }

    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        if (!mype) {
            int nelems;
            nelems = size / sizeof(int);

            latency<<<1, 1>>>(data_d, flag_d, nelems, mype, iter, skip);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        printf("   Put with warp \n");
        printf("   size(bytes) \t latency(us)\n");
        fflush(stdout);
    }

    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        if (!mype) {
            int nelems;
            nelems = size / sizeof(int);

            latency_warp<<<1, THREADS_PER_WARP>>>(data_d, flag_d, nelems, mype, iter, skip);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        printf("   Put with block \n");
        printf("   size(bytes) \t latency(us)\n");
        fflush(stdout);
    }

    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        if (!mype) {
            int nelems;
            nelems = size / sizeof(int);

            latency_block<<<1, THREADS_PER_BLOCK>>>(data_d, flag_d, nelems, mype, iter, skip);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }

        nvshmem_barrier_all();
    }

finalize:

    if (data_d) nvshmem_free(data_d);
    if (flag_d) nvshmem_free(flag_d);

    finalize_wrapper();

    return 0;
}
