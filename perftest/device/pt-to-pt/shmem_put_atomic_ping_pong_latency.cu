#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define MAX_MSG_SIZE 1 * 1024 * 1024
#define UNROLL 8

__global__ void ping_pong(volatile int *data_d, uint64_t *flag_d,
                          int len, int pe, int iter, int skip, int *hflag) {
    long long int start, stop;
    double usec, time;
    int i, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < (iter + skip); i++) {
        if (i == skip) start = clock64();

        if (pe) {
            nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));

            nvshmem_int_put_nbi((int *)data_d, (int *)data_d, len, peer);

            nvshmem_fence();

            nvshmem_uint64_atomic_inc(flag_d, peer);
        } else {
            nvshmem_int_put_nbi((int *)data_d, (int *)data_d, len, peer);

            nvshmem_fence();

            nvshmem_uint64_atomic_inc(flag_d, peer);

            nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));
        }
    }
    stop = clock64();
    nvshmem_quiet();
    *hflag = 1;

    if ((pe == 0) && !tid) {
        time = (stop - start) / iter;
        usec = time * 1000 / clockrate;
        printf("%7lu \t %8.2f \n", len * sizeof(int), usec);
    }
}

int main(int c, char *v[]) {
    int mype, npes, size;
    uint64_t *flag_d = NULL;
    int *data_d = NULL;
    hipStream_t stream;

    int iter = 500;
    int skip = 50;
    int max_msg_size = MAX_MSG_SIZE;

    init_wrapper(&c, &v);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    data_d = (int *)nvshmem_malloc(max_msg_size);
    flag_d = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
    CUDA_CHECK(hipMemset(data_d, 0, max_msg_size));
    CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));

    int *hflag, *hflag_d;
    CUDA_CHECK(hipHostAlloc((void **)&hflag, sizeof(uint64_t), 0));
    *hflag = 0;
    CUDA_CHECK(hipHostGetDevicePointer(&hflag_d, hflag, 0));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    if (mype == 0) {
        printf("Note: This test measures full round-trip latency\n");
        printf("   size(bytes) \t latency(us)\n");
        fflush(stdout);
    }

    for (size = sizeof(int); size <= max_msg_size; size *= 2) {
        int nelems, status = 0;
        nelems = size / sizeof(int);
        void *args[] = {&data_d, &flag_d, &nelems, &mype, &iter, &skip, &hflag_d};

        CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        CUDA_CHECK(hipDeviceSynchronize());
        nvshmem_barrier_all();

        *hflag = 0;
        status = nvshmemx_collective_launch((const void *)ping_pong, 1, 1, args, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }

        while (*((volatile int *)hflag) != 1)
            ;

        nvshmem_barrier_all();
    }

    CUDA_CHECK(hipDeviceSynchronize());

finalize:

    if (data_d) nvshmem_free(data_d);
    if (flag_d) nvshmem_free(flag_d);

    finalize_wrapper();

    return 0;
}
