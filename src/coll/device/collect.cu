#include "hip/hip_runtime.h"
/*
 * * Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 * *
 * * See COPYRIGHT for license information
 * */

#include "nvshmem.h"
#include "gpu_coll.h"

#ifdef __CUDA_ARCH__

#define NVSHMEMI_GPU_COLLECT_P2P_ALL_PUSH(SUFFIX, dest, source, nelems, PE_start, logPE_stride, \
                                          PE_size, pSync)                                       \
    do {                                                                                        \
        int stride = 1 << logPE_stride;                                                         \
        int next_rank;                                                                          \
        int next_offset;                                                                        \
        int mype = nvshmemi_mype_d;                                                              \
                                                                                                \
        for (int ii = 0; ii < PE_size; ii++) {                                                  \
            next_rank = (mype + (ii * stride)) % (stride * PE_size);                            \
            next_offset = nelems * ((mype - PE_start) / stride);                                \
            /*XXX:typecast dest to suppress warning "arithmetic on pointer to void or function  \
             * typ"*/                                                                           \
            nvshmem_put##SUFFIX##_nbi((uint##SUFFIX##_t *)dest + next_offset, source, nelems,   \
                                      next_rank);                                               \
        }                                                                                       \
        nvshmem_barrier(PE_start, logPE_stride, PE_size, pSync);                                \
    } while (0)

#define NVSHMEMI_GPU_COLLECT_ALL_PUSH(SUFFIX, dest, source, nelems, PE_start, logPE_stride,      \
                                      PE_size, pSync)                                            \
    do {                                                                                         \
        int stride = 1 << logPE_stride;                                                          \
        int next_rank;                                                                           \
        int next_offset;                                                                         \
        int mype = nvshmemi_mype_d;                                                               \
        int offset;                                                                              \
        char *round_dest;                                                                        \
        offset =                                                                                 \
            (char *)dest - (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + \
                                          nvshmemi_mype_d));                                      \
                                                                                                 \
        for (int ii = 0; ii < PE_size; ii++) {                                                   \
            next_rank = (mype + (ii * stride)) % (stride * PE_size);                             \
            next_offset = nelems * ((mype - PE_start) / stride);                                 \
            round_dest = (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d +   \
                                        next_rank)) +                                            \
                         offset + sizeof(uint##SUFFIX##_t) * next_offset;                        \
            GPU_BITS_COPY_DIRECT(SUFFIX, (uint##SUFFIX##_t *)round_dest, source, nelems);        \
        }                                                                                        \
        nvshmem_barrier(PE_start, logPE_stride, PE_size, pSync);                                 \
    } while (0)

#ifdef NVSHMEM_GPU_COLL_USE_LDST
#define NVSHMEMI_GPU_COLLECT(SUFFIX, dest, source, nelems, PE_start, logPE_stride, PE_size, pSync) \
    do {                                                                                           \
        NVSHMEMI_GPU_COLLECT_ALL_PUSH(SUFFIX, dest, source, nelems, PE_start, logPE_stride,        \
                                      PE_size, pSync);                                             \
    } while (0)
#else
#define NVSHMEMI_GPU_COLLECT(SUFFIX, dest, source, nelems, PE_start, logPE_stride, PE_size, pSync) \
    do {                                                                                           \
        NVSHMEMI_GPU_COLLECT_P2P_ALL_PUSH(SUFFIX, dest, source, nelems, PE_start, logPE_stride,    \
                                          PE_size, pSync);                                         \
    } while (0)
#endif

#define DEFN_NVSHMEM_GPU_COLLECT(SUFFIX)                                                    \
    __device__ void nvshmem_collect##SUFFIX(void *dest, const void *source, size_t nelems,  \
                                            int PE_start, int logPE_stride, int PE_size,    \
                                            long *pSync) {                                  \
        NVSHMEMI_GPU_COLLECT(SUFFIX, dest, source, nelems, PE_start, logPE_stride, PE_size, \
                             pSync);                                                        \
    }

#define DEFN_NVSHMEM_GPU_COLLECT_TYPES() \
    DEFN_NVSHMEM_GPU_COLLECT(32);        \
    DEFN_NVSHMEM_GPU_COLLECT(64);

DEFN_NVSHMEM_GPU_COLLECT_TYPES();

#endif
