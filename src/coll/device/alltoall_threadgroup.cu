#include "hip/hip_runtime.h"
/*
 * * Copyright (c) 2017-2018, NVIDIA CORPORATION. All rights reserved.
 * *
 * * See COPYRIGHT for license information
 * */

#include "nvshmem.h"
#include "nvshmemx.h"
#include "gpu_coll.h"
#include <cstdio>
#include <cassert>

#ifdef __CUDA_ARCH__

#define NVSHMEMI_GPU_ALLTOALL_THREADGROUP_P2P_ALLPUSH(SC, SUFFIX, dest, source, nelems, PE_start, \
                                                      logPE_stride, PE_size, pSync)               \
    do {                                                                                          \
        int stride = 1 << logPE_stride;                                                           \
        int next_rank;                                                                            \
        int src_offset;                                                                           \
        int dst_offset;                                                                           \
        int mype = nvshmemi_mype_d;                                                                \
                                                                                                  \
        for (int ii = 0; ii < PE_size; ii++) {                                                    \
            next_rank = (mype + (ii * stride)) % (stride * PE_size);                              \
            src_offset = nelems * ((next_rank - PE_start) / stride);                              \
            dst_offset = nelems * ((mype - PE_start) / stride);                                   \
            nvshmemx_put##SUFFIX##_nbi_##SC((uint##SUFFIX##_t *)dest + dst_offset,                \
                                            (uint##SUFFIX##_t *)source + src_offset, nelems,      \
                                            next_rank);                                           \
        }                                                                                         \
        nvshmemx_barrier_##SC(PE_start, logPE_stride, PE_size, pSync);                            \
    } while (0)

#define NVSHMEMI_GPU_ALLTOALL_THREADGROUP_ALLPUSH(SC, SUFFIX, dest, source, nelems, PE_start,    \
                                                  logPE_stride, PE_size, pSync)                  \
    do {                                                                                         \
        int stride = 1 << logPE_stride;                                                          \
        int next_rank;                                                                           \
        int src_offset;                                                                          \
        int dst_offset;                                                                          \
        int mype = nvshmemi_mype_d;                                                               \
        int offset;                                                                              \
        char *round_dest;                                                                        \
        NVSHMEMI_DECL_THREAD_IDX_##SC();                                                         \
        NVSHMEMI_DECL_THREADGROUP_SIZE_##SC();                                                   \
        offset =                                                                                 \
            (char *)dest - (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d + \
                                          nvshmemi_mype_d));                                      \
                                                                                                 \
        for (int ii = 0; ii < PE_size; ii++) {                                                   \
            next_rank = (mype + (ii * stride)) % (stride * PE_size);                             \
            src_offset = nelems * ((next_rank - PE_start) / stride);                             \
            dst_offset = nelems * ((mype - PE_start) / stride);                                  \
            round_dest = (char *)(__ldg((const long long unsigned *)nvshmemi_peer_heap_base_d +   \
                                        next_rank)) +                                            \
                         offset + sizeof(uint##SUFFIX##_t) * dst_offset;                         \
            GPU_BITS_COPY_THREADGROUP_DIRECT(SUFFIX, (uint##SUFFIX##_t *)round_dest,             \
                                             (uint##SUFFIX##_t *)source + src_offset, nelems,    \
                                             myIdx, groupSize);                                  \
        }                                                                                        \
        nvshmemx_barrier_##SC(PE_start, logPE_stride, PE_size, pSync);                           \
    } while (0)

#ifdef NVSHMEM_GPU_COLL_USE_LDST
#define NVSHMEMI_GPU_ALLTOALL_THREADGROUP(SC, SUFFIX, dest, source, nelems, PE_start,         \
                                          logPE_stride, PE_size, pSync)                       \
    do {                                                                                      \
        NVSHMEMI_GPU_ALLTOALL_THREADGROUP_ALLPUSH(SC, SUFFIX, dest, source, nelems, PE_start, \
                                                  logPE_stride, PE_size, pSync);              \
    } while (0)
#else
#define NVSHMEMI_GPU_ALLTOALL_THREADGROUP(SC, SUFFIX, dest, source, nelems, PE_start,             \
                                          logPE_stride, PE_size, pSync)                           \
    do {                                                                                          \
        NVSHMEMI_GPU_ALLTOALL_THREADGROUP_P2P_ALLPUSH(SC, SUFFIX, dest, source, nelems, PE_start, \
                                                      logPE_stride, PE_size, pSync);              \
    } while (0)
#endif

#define DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP(SC, SUFFIX)                                         \
    __device__ void nvshmemx_alltoall##SUFFIX##_##SC(void *dest, const void *source,               \
                                                     size_t nelems, int PE_start,                  \
                                                     int logPE_stride, int PE_size, long *pSync) { \
        NVSHMEMI_GPU_ALLTOALL_THREADGROUP(SC, SUFFIX, dest, source, nelems, PE_start,              \
                                          logPE_stride, PE_size, pSync);                           \
    }

#define DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP_TYPES(SC) \
    DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP(SC, 8);       \
    DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP(SC, 16);      \
    DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP(SC, 32);      \
    DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP(SC, 64);

DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP_TYPES(warp);
DEFN_NVSHMEMX_GPU_ALLTOALL_THREADGROUP_TYPES(block);

#endif

#define ALLTOALL_ON_STREAM_KERNEL(BITS)                                                            \
    __global__ void alltoall##BITS##_on_stream_kernel(                                             \
        void *dest, const void *source, size_t nelems, int PE_start, int logPE_stride,             \
        int PE_size, long *pSync) {                                                                \
        if (!blockIdx.x)                                                                           \
            nvshmemx_alltoall##BITS##_block(dest, source, nelems, PE_start, logPE_stride, PE_size, \
                                            pSync);                                                \
    }

ALLTOALL_ON_STREAM_KERNEL(32);
ALLTOALL_ON_STREAM_KERNEL(64);

#define CALL_ALLTOALL_ON_STREAM(BITS)                                                              \
    extern "C" void call_alltoall##BITS##_on_stream_kern(                                          \
        void *dest, const void *source, size_t nelems, int PE_start, int logPE_stride,             \
        int PE_size, long *pSync, hipStream_t stream) {                                           \
        int num_threads_per_block = (MAX_THREADS_PER_CTA > nelems) ? nelems : MAX_THREADS_PER_CTA; \
        int num_blocks = 1;                                                                        \
        alltoall##BITS##_on_stream_kernel<<<num_blocks, num_threads_per_block, 0, stream>>>(       \
            dest, source, nelems, PE_start, logPE_stride, PE_size, pSync);                         \
        CUDA_RUNTIME_CHECK(hipGetLastError());                                                    \
    }

CALL_ALLTOALL_ON_STREAM(32);
CALL_ALLTOALL_ON_STREAM(64);
